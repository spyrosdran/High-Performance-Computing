# include <stdlib.h>
# include <stdio.h>
# include <time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024

__global__ void multiply(int *a, int *b, int *c, int N){

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int j;

    if(index<N)
        for (j = 0; j < N; j++)
            c[index] += a[index*j] * b[j];

}

int main ( int argc, char *argv[] ) {
	
	int *a, *b, *c, *da, *db, *dc;
	int i, j, N;
    hipEvent_t start, stop, startCopyHostToDevice, endCopyHostToDevice, startCopyDeviceToHost, endCopyDeviceToHost;
    float total_time = 0, copyHostToDevice_time = 0, copyDeviceToHost_time = 0;
 
	if (argc != 2) { printf ("Usage : %s <matrix size>\n", argv[0]); exit(1);}

	// Retrieving matrix size
	N = strtol(argv[1], NULL, 10);

	//Allocate memory for the matrices
    a = (int*) malloc (N * N * sizeof(int));	
	b = (int*) malloc (N * sizeof(int));
	c = (int*) malloc (N * sizeof(int));

	// Assign values to the matrices
	srand(time(NULL));

	for (i = 0; i < N * N; i++)
        a[i] = rand()%10;

	for ( i = 0; i < N; i++ ) {
	    b[i] = rand()%10;
		c[i] = 0;
    }

    // Create events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startCopyHostToDevice);
    hipEventCreate(&endCopyHostToDevice);
    hipEventCreate(&startCopyDeviceToHost);
    hipEventCreate(&endCopyDeviceToHost);

    // Start timing
    hipEventRecord(start);

    // Allocate memory for the device variables
    hipMalloc((void **)&da, N * N * sizeof(int));
    hipMalloc((void **)&db, N * sizeof(int));
    hipMalloc((void **)&dc, N * sizeof(int));

    // Copy variables from host to device
    hipEventRecord(startCopyHostToDevice);

    hipMemcpy(da, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dc, c, N * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(endCopyHostToDevice);
    hipEventSynchronize(endCopyHostToDevice);

	// Multiplication
	multiply<<<((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(da, db, dc, N);

    hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));

    // Copy the result vector from device to host
    hipEventRecord(startCopyDeviceToHost);

    hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(endCopyDeviceToHost);
    hipEventSynchronize(endCopyDeviceToHost);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);    

    // Compute times
    hipEventElapsedTime(&total_time, start, stop);
    hipEventElapsedTime(&copyHostToDevice_time, startCopyHostToDevice, endCopyHostToDevice);
    hipEventElapsedTime(&copyDeviceToHost_time, startCopyDeviceToHost, endCopyDeviceToHost);

    // Free the unused memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);

	/* // Print the computed matrices
	for ( i = 0; i < N; i++ ) {
		for ( j = 0; j < N; j++ )
			printf ("%d ", a[i*j]); 
		printf("\t %d ", b[i]);
		printf("\t %d \n", c[i]);
	} */

    free(a);
    free(b);
    free(c);

    printf("Total time elapsed: %fs\n", total_time / 1000);
    printf("Copy host to device time: %fs\n", copyHostToDevice_time / 1000);
    printf("Copy device to host time: %fs\n", copyDeviceToHost_time / 1000);

    return 0;
}



